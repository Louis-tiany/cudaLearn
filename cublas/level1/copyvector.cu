#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define N 6

//compute alpha*x+y,x and y are vector,alpha is a scalar 
int main(void)
{
	hipError_t cudaStatus;
	hipblasStatus_t status;
	hipblasHandle_t handle;

	float *x;
	x = (float *)malloc(N * sizeof(*x));
	for (int i = 0; i < N; ++i)
		x[i] = float(i);
	
	printf("x:");
	for (int i = 0; i < N; ++i)
		printf("%4.0f", x[i]);
	printf("\n");

	float *y;
	y = (float *)malloc(N * sizeof(*x));


	//device codes

	float *dx, *dy;
	cudaStatus = hipMalloc((void **)dx, N * sizeof(*x));
	cudaStatus = hipMalloc((void **)dy, N * sizeof(*y));

	status = hipblasCreate(&handle);//create context
	status = hipblasSetVector(N, sizeof(*x), x, 1, dx, 1);//x->dx

	//copy values:dx--->dy
	status = hipblasScopy(handle, N, dx, 1, dy, 1);

	//copy device values to host values: dy --->y
	status = hipblasSetVector(N, sizeof(float), dy, 1, y, 1);

	printf("after copy values\n");
	for (int i = 0; i < N; ++i)
		printf("%4.0f", y[i]);
	printf("\n");


	//free device memory
	hipFree(dx);
	hipFree(dy);
	
	//free cuda context
	hipblasDestroy(handle);

	//free host memory
	free(x);
	free(y);


	return 0;
}

