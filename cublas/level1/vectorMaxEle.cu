#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define N 6


int main(void)
{
	hipError_t cudaStatus;
	hipblasStatus_t status;
	hipblasHandle_t handle;

	float *vec;
	vec = (float *)malloc(N * sizeof(float));
	for (int i = 0; i < N; ++i)
		vec[i] = (float)i;
	printf("x:\n");
	for (int i = 0; i < N; ++i)
		printf("%4.0f",vec[i]);
	printf("\n");

	//device codes
	float *dVec;
	cudaStatus = hipMalloc((void **)&dVec, N * sizeof(float));
	status = hipblasCreate(&handle);
	status = hipblasSetVector(N, sizeof(*vec), vec, 1, dVec, 1);

	int result;//note max value's index
	status = hipblasIsamax(handle, N, dVec, 1, &result);
	
	printf(" m a x | vec [ i ] | : %4 . 0 f \ n",fabs(vec[result-1]));
	
	//free device memory
	hipFree(dVec);
	//destory cublas context
	hipblasDestroy(handle);

	//free host memory 
	free(vec);

	return 0;
}
