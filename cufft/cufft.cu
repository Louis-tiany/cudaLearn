#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipfft/hipfft.h"
#include <iostream>


void checkError(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		std::cout << "line number:" << __LINE__ << "\n";
		std::cout << "error:" << hipGetErrorString(cudaStatus) << "\n";
	}
}

void fftStatus(hipfftResult cufftStatus)
{
	if (cufftStatus != HIPFFT_SUCCESS)
	{
		std::cout << "line number:" << __LINE__ << "\n";
	}
}

int main()
{
	const int Length = 10;
	hipfftComplex *signalIn,*signalOut;
	signalIn = (hipfftComplex *)malloc(Length * sizeof(hipfftComplex));
	signalOut= (hipfftComplex *)malloc(Length * sizeof(hipfftComplex));
	for (int i = 0; i < Length; ++i)
	{
		signalIn[i].x = (float)i;
		signalIn[i].y = 2.0f;
	}

	printf("data to be tranforded:\n");
	for (int i = 0; i < Length; ++i)
		printf("%f+j%f\n", signalIn[i].x, signalIn[i].y);
	printf("\n");

	//alloc memory in device
	hipfftComplex *dSignalIn,*dSignalOut;
	hipMalloc((void **)&dSignalIn, Length * sizeof(hipfftComplex));
	hipMalloc((void **)&dSignalOut, Length * sizeof(hipfftComplex));

	//copy memory from to host
	hipMemcpy(dSignalIn, signalIn, Length * sizeof(hipfftComplex), hipMemcpyHostToDevice);


	//create handle of forward transform and inverse transform
	hipfftHandle fftHandle, fftInverseHandle;
	fftStatus(hipfftPlan1d(&fftHandle, Length, HIPFFT_C2C, 1));
	fftStatus(hipfftPlan1d(&fftInverseHandle, Length, HIPFFT_C2C, 1));

	hipfftExecC2C(fftHandle, dSignalIn, dSignalOut, HIPFFT_FORWARD);
	//copy memory from device to host
	checkError(hipMemcpy(signalOut, dSignalOut, Length * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	printf("\ntransform result:\n");
	for (int i = 0; i < Length; ++i)
		printf("%f+j%f\n", signalOut[i].x, signalOut[i].y);
	printf("\n");

	//inverse transform
	hipfftExecC2C(fftInverseHandle, dSignalOut, dSignalIn, HIPFFT_BACKWARD);
	checkError(hipMemcpy(signalIn, dSignalIn, Length * sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	printf("\ninverse transform result:\n");
	for (int i = 0; i < Length; ++i)
		printf("%f+j%f\n", signalIn[i].x, signalIn[i].y);
	printf("\n");

	hipFree(signalIn);
	hipFree(signalOut);
	hipfftDestroy(fftHandle);
	hipfftDestroy(fftInverseHandle);
	free(signalIn);
	free(signalOut);

	return 0;
}
