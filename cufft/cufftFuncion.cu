#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include""
#include "hipblas.h"
#include "hipfft/hipfft.h"
#include <iostream>
#include <hip/hip_runtime.h>

void checkError(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		std::cout << "line number:" << __LINE__ << "\n";
		std::cout << "error:" << hipGetErrorString(cudaStatus) << "\n";
	}
}

void fftStatus(hipfftResult cufftStatus)
{
	if (cufftStatus != HIPFFT_SUCCESS)
	{
		std::cout << "line number:" << __LINE__ << "\n";
	}
}



void fft1D(hipComplex *data,hipComplex *result,int num)
{
	int Length = num;
	hipComplex *dData;
	hipComplex *dResult;
	hipMalloc((void **)&dData, Length * sizeof(float));
	hipMalloc((void **)&dResult, Length * sizeof(hipComplex));
	//create handle of fft real--->complex
	hipfftHandle fftHandle;
	fftStatus(hipfftPlan1d(&fftHandle, Length, HIPFFT_C2C, 1));

	//send data to device
	hipMemcpy(dData, data, Length * sizeof(float), hipMemcpyHostToDevice);

	hipfftExecC2C(fftHandle, dData, dResult,HIPFFT_FORWARD);
	//copy memory from device to host
	checkError(hipMemcpy(result, dResult, Length*sizeof(hipfftComplex), hipMemcpyDeviceToHost));

	hipfftDestroy(fftHandle);
	hipFree(dData);
	hipFree(dResult);
}

//use for ifft function
__global__ void fftNorm(hipComplex *data,int length)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	data[idx].x /= length;
	data[idx].y /= length;
}


void fftInverse1D(hipComplex *data,hipComplex *result,int num)
{
	int Length = num;
	hipComplex * dData;
	hipComplex *dResult;
	hipfftHandle fftInverseHandle;
	fftStatus(hipfftPlan1d(&fftInverseHandle, Length, HIPFFT_C2C, 1));

	hipMalloc((void **)&dData, Length * sizeof(hipComplex));
	hipMalloc((void **)&dResult, Length  * sizeof(hipComplex));

	//send data
	checkError(hipMemcpy(dData, data, Length * sizeof(hipComplex), hipMemcpyHostToDevice));
	//compute
	fftStatus(hipfftExecC2C(fftInverseHandle, dData, dResult,HIPFFT_BACKWARD));
	//normlizing
	dim3 grid(8);
	dim3 block(Length / 8);
	fftNorm << <grid, block >> > (dResult, Length);
	//fetch data
	checkError(hipMemcpy(result, dResult, Length * sizeof(float), hipMemcpyDeviceToHost));

	hipfftDestroy(fftInverseHandle);
	hipFree(dData);
	hipFree(dResult);
}



//get length of complex array's abs
//input *src and *dest are device memory's pointer
__global__ void complexAbs(hipComplex *src, float *dest)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	dest[idx] = hipCabsf(src[idx]);
}

//src and dest are host memory's pointer
void complexNorm(hipComplex *src, float *dest, int num)
{
	int Length = num;
	//the size of gird and block should be changed depending input data
	dim3 grid(8);
	dim3 block(Length / 8);

	hipfftComplex *dSrc;
	float *dDest;
	hipMalloc((void **)&dSrc, Length * sizeof(hipfftComplex));
	hipMalloc((void **)&dDest, Length * sizeof(float));
	//send data
	checkError(hipMemcpy(dSrc, src, Length * sizeof(hipfftComplex), hipMemcpyHostToDevice));
	//compute
	complexAbs << <grid, block >> > (dSrc, dDest);
	//fetch data
	checkError(hipMemcpy(dest, dDest, Length * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(dSrc);
	hipFree(dDest);
}

__global__ void complexConjUtil(hipComplex *src, hipComplex *dest)
{
	int idx = blockDim.x*blockIdx.x + threadIdx.x;
	dest[idx].x = src[idx].x;
	dest[idx].y = -src[idx].y;
}

//src and dest is host's memory pointer
void complexConj(hipComplex *src, hipComplex *dest,int num)
{

	hipComplex *dSrc, *dDest;

	hipMalloc((void **)&dSrc, num * sizeof(hipComplex));
	hipMalloc((void **)&dDest, num * sizeof(hipComplex));

	int Length = num;
	dim3 grid(8);
	dim3 block(Length / 8);
	//send data
	checkError(hipMemcpy(dSrc, src, Length * sizeof(hipfftComplex), hipMemcpyHostToDevice));
	//compute
	complexConjUtil << <grid, block >> > (dSrc, dDest);
	//fetch data
	checkError(hipMemcpy(dest, dDest, Length * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
	hipFree(dSrc);
	hipFree(dDest);
}


int main()
{

	const int Length = 16;

	hipComplex *data;
	data = (hipComplex *)malloc(Length * sizeof(hipComplex));
	for (int i = 0; i < Length; ++i)
	{
		data[i].x = (float)i;
		data[i].y = (float)i;
	}
	printf("data:\n");
	for (int i = 0; i < Length; ++i)
		printf("%f+j%f\n", data[i].x, data[i].y);
	printf("\n");


	hipComplex *dataConj;
	dataConj = (hipComplex *)malloc(Length * sizeof(hipComplex));
	complexConj(data, dataConj, Length);
	printf("data's Conjugate:\n");
	for (int i = 0; i < Length; ++i)
		printf("%f+j%f\n", dataConj[i].x, dataConj[i].y);
	printf("\n");
	free(dataConj);



	hipComplex *result;
	result = (hipComplex *)malloc(Length * sizeof(hipComplex));

	fft1D(data, result, Length);

	printf("fft result:\n");
	for (int i = 0; i < Length; ++i)
		printf("%f+j%f\n", result[i].x, result[i].y);
	printf("\n");

	fftInverse1D(result, data, Length);
	printf("inverse fft result:\n");
	for (int i = 0; i < (Length); ++i)
		printf("%f+j%f\n", data[i].x,data[i].y);
	printf("\n");

	free(data);
	free(result);

	printf("\n-------------------------------\n");



	return 0;
}

